
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#define STORAGE_SIZE 1085440
#define MAX_FILE_SIZE 1048576

#define BLOCK_SIZE 1024
#define INODE_START 2
#define INODE_SIZE 28
#define INODE_COUNT 1024
#define INODE_LOC(i) (INODE_START + i*INODE_SIZE)
#define DATA_START 36864
#define ERROR 65535

#define DATAFILE "data.bin"
#define OUTFILE "snapshot.bin"
#define G_WRITE 0
#define G_READ 1
#define WRITE_SUCCESS 0
#define WRITE_ERROR 1
#define READ_SUCCESS 0
#define READ_ERROR 1

#define NAME_LENGTH 21

#define LS_D 0
#define LS_S 1
#define RM 2

#define TIME_LOC DATA_START-2
#define TIME (read2bytes(TIME_LOC))

typedef unsigned char uchar;
typedef uint32_t u32;

__device__ __managed__ uchar *volume;

int load_binaryFile(const char *filename, uchar *a, int max_size);
void write_binaryFIle(const char *filename, uchar *a, int size);
__device__ u32 open(const char *name, uchar mode);
__device__ uchar write(uchar *input, int n, uchar fp);
__device__ uchar read(uchar *output, int n, uchar fp);
__device__ void gsys(uchar arg, const char* file=NULL);
__device__ void init_volume();

__device__ int my_strcmp(uchar *a, const char *b) {
	for(int i = 0; i < NAME_LENGTH; i++) {
		if(a[i] == 0 && b[i] == 0) return 0;
		else if(a[i] != b[i]) return 1;
	}
	return 1;
}
__device__ void my_strcpy(uchar *d, const char *s) {
	int i;
	for(i = 0; i < NAME_LENGTH-1; i++) {
		d[i] = s[i];
		if(s[i] == 0) return;
	}
	d[i] = 0;
}
__device__ int read2bytes(int i) {
	return (volume[i]<<8) + volume[i+1];
}
__device__ void write2bytes(int num, int i) {
	volume[i+1] = num;
	volume[i] = (num>>8);
}
__device__ int find_room() {
	int i = read2bytes(0);
	int cur = INODE_LOC(i);
	int j = read2bytes(cur+1);
	cur = INODE_LOC(j);
	int k = read2bytes(cur+1);
	write2bytes(k, 0);
	return j;
}
__device__ void free_room(int v) {
	int i = read2bytes(0);
	int cur = INODE_LOC(v);
	volume[cur] = 0; //set to empty
	write2bytes(v, 0);
	write2bytes(i, cur+1);
}

__global__ void mykernel(uchar *input, uchar *output) {
	init_volume();
	//####kernel start####
	u32 fpa = open("a.txt\0", G_WRITE);
	write(input, 30, fpa);
	u32 fpb = open("b.txt\0", G_WRITE);
	write(input, 10, fpa);
	gsys(LS_S);
	gsys(RM, "a.txt\0");
	gsys(LS_D);
	read(output, 5, fpa);
	//####kernel end####
}
int main() {
	hipSetDevice(3);
	hipMallocManaged(&volume, STORAGE_SIZE);

	uchar *input, *output;
	hipMallocManaged(&input, MAX_FILE_SIZE);
	hipMallocManaged(&output, MAX_FILE_SIZE);
	for(int i = 0; i < MAX_FILE_SIZE; i++) output[i] = 0;

	load_binaryFile(DATAFILE, input, MAX_FILE_SIZE);

	mykernel<<<1, 1>>>(input, output);
	hipDeviceSynchronize();
	write_binaryFIle(OUTFILE, output, MAX_FILE_SIZE);
	hipDeviceReset();

	return 0;
}

int load_binaryFile(const char *filename, uchar *a, int max_size) {
	FILE *fp = fopen(filename, "rb");
	int i = 0;
	while(!feof(fp) && i < max_size) {
		fread(a+i, sizeof(uchar), 1, fp);
		i++;
	}
	return i;
}
void write_binaryFIle(const char *filename, uchar *a, int size) {
	FILE *fp = fopen(filename, "wb+");
	fwrite(a, sizeof(uchar), size, fp);
}
__device__ void init_volume() {
	write2bytes(0, TIME_LOC); //time init
	for(int i = 0; i < INODE_COUNT; i++) {
		int cur = INODE_LOC(i);
		volume[cur] = 0; //set empty
		write2bytes(i+1, cur+1); //point to i+1
	}
}
__device__ u32 open(const char *name, uchar mode) {
	for(int i = 0; i < INODE_COUNT; i++) {
		int cur = INODE_LOC(i);
		if(my_strcmp(volume+cur+7, name) == 0) {
			write2bytes(0, cur+1); //set fp to 0
			return i;
		}
	}
	if(mode == G_WRITE) { //create
		int i = find_room();
		if(i == -1) return ERROR;
		int cur = INODE_LOC(i);
		volume[cur] = 1; //set not empty
		write2bytes(0, cur+1); //set fp to 0
		write2bytes(0, cur+3); //set size to 0
		int time = TIME;
		write2bytes(time, cur+5); //set timestamp
		write2bytes(time+1, TIME_LOC); //increase time
		my_strcpy(volume+cur+7, name); //set name
		return i;
	}
	else return ERROR;
}

__device__ uchar write(uchar *input, int n, uchar fp) {
	if(fp == ERROR) return WRITE_ERROR;
	if(n > BLOCK_SIZE) return WRITE_ERROR;
	int cur = INODE_LOC(fp);
	int cur_block = DATA_START + fp*BLOCK_SIZE;
	write2bytes(n, cur+3); //change file size
	int time = TIME;
	write2bytes(time, cur+5); //set time stamp
	write2bytes(time+1, TIME_LOC); //increase time
	for(int i = 0; i < n; i++) {
		volume[cur_block+i] = input[i];
	}
	return WRITE_SUCCESS;
}
__device__ uchar read(uchar *output, int n, uchar fp) {
	if(fp == ERROR) return READ_ERROR;
	if(n > BLOCK_SIZE) return READ_ERROR;
	int cur_block = DATA_START + fp*BLOCK_SIZE;
	for(int i = 0; i < n; i++) {
		output[i] = volume[cur_block+i];
	}
	return READ_SUCCESS;
}
__device__ void gsys(uchar arg, const char *file) {
	if(arg == LS_S) {
		printf("===sorted by file size===\n");
		int a[INODE_COUNT];
		int n = 0;
		for(int i = 0; i < INODE_COUNT; i++) {
			int cur = INODE_LOC(i);
			if(volume[cur]) a[n++] = cur; //inode not empty
		}
		for(int i = 0; i < n; i++) {
			for(int j = i+1; j < n; j++) {
				if(read2bytes(a[i]+3) > read2bytes(a[j]+3)) {
					int tmp = a[i];
					a[i] = a[j];
					a[j] = tmp;
				}
			}
		}
		for(int i = 0; i < n; i++) printf("%s %d\n", volume+a[i]+7, read2bytes(a[i]+3));
	}
	else if(arg == LS_D) {
		printf("===sorted by modified time===\n");
		int a[INODE_COUNT];
		int n = 0;
		for(int i = 0; i < INODE_COUNT; i++) {
			int cur = INODE_LOC(i);
			if(volume[cur]) a[n++] = cur; //inode not empty
		}
		for(int i = 0; i < n; i++) {
			for(int j = i+1; j < n; j++) {
				if(read2bytes(a[i]+5) < read2bytes(a[j]+5)) {
					int tmp = a[i];
					a[i] = a[j];
					a[j] = tmp;
				}
			}
		}
		for(int i = 0; i < n; i++) printf("%s\n", volume+a[i]+7);
	}
	else if(arg == RM) {
		int i;
		for(i = 0; i < INODE_COUNT; i++) {
			int cur = INODE_LOC(i);
			if(volume[cur]) { //not empty
				if(my_strcmp(volume+cur+7, file) == 0) break;
			}
		}
		if(i == INODE_COUNT) printf("No such file %s!\n", file);
		else {
			free_room(i);
		}
	}
}
